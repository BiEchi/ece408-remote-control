#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "CUDA error: ", hipGetErrorString(err));              \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      return -1;                                                          \
    }                                                                     \
  } while (0)

//@@ Define any useful program-wide constants here
#define TILE_WIDTH 3
#define MASK_WIDTH 3
#define MASK_RADIUS 1
#define BLOCK_WIDTH 3+(3-1) // TILE_WIDTH+(MASK_WIDTH-1)

//@@ Define constant memory for device kernel here
__constant__ float deviceKernel[MASK_WIDTH][MASK_WIDTH][MASK_WIDTH];

//@@ helper functions
__device__ int io_inside_range(int x_size, int y_size, int z_size, int x, int y, int z)
{
  if (x>=0 && y>=0 && z>=0 && x<x_size && y<y_size && z<z_size) return 1;
  else return 0;
}

__device__ int tile_inside_range(int thr_x, int thr_y, int thr_z)
{
  if (thr_x < TILE_WIDTH && thr_y < TILE_WIDTH && thr_z < TILE_WIDTH) return 1; 
  else return 0;
}

__device__ int flatten(int x_size, int y_size, int z_size, int x, int y, int z)
{
  return (z * x_size * y_size + y * x_size + x);
}

// GPU kernel function
__global__ void conv3d(float *input, float *output, const int z_size, const int y_size, const int x_size) {
  //@@ Insert kernel code here
  
  // define local variables for acceleration
  int x_out = blockIdx.x * TILE_WIDTH + threadIdx.x;
  int y_out = blockIdx.y * TILE_WIDTH + threadIdx.y;
  int z_out = blockIdx.z * TILE_WIDTH + threadIdx.z;
  int x_in = x_out - MASK_RADIUS;
  int y_in = y_out - MASK_RADIUS;
  int z_in = z_out - MASK_RADIUS;

  // construct tiled memory
  __shared__ float N_ds[BLOCK_WIDTH][BLOCK_WIDTH][BLOCK_WIDTH];
  if (io_inside_range(x_size, y_size, z_size, x_in, y_in, z_in))
    N_ds[threadIdx.z][threadIdx.y][threadIdx.x] = input[flatten(x_size, y_size, z_size, x_in, y_in, z_in)];
  else N_ds[threadIdx.z][threadIdx.y][threadIdx.x] = (float)0;

  __syncthreads();

  float element = 0;
  //if (tile_inside_range(threadIdx.x, threadIdx.y, threadIdx.z))
  if ((threadIdx.x < TILE_WIDTH) && (threadIdx.y < TILE_WIDTH) && (threadIdx.z < TILE_WIDTH))
  {
    for (int z_incre = 0; z_incre < MASK_WIDTH; z_incre++)
      for (int y_incre = 0; y_incre < MASK_WIDTH; y_incre++)
        for (int x_incre = 0; x_incre < MASK_WIDTH; x_incre++)
          // apply the convolution calculation
          element += deviceKernel[z_incre][y_incre][x_incre] * N_ds[z_incre+threadIdx.z][y_incre+threadIdx.y][x_incre+threadIdx.x];
    if (io_inside_range(x_size, y_size, z_size, x_out, y_out, z_out))
      output[flatten(x_size, y_size, z_size, x_out, y_out, z_out)] = element;
  }
  return;
}

// main function
int main(int argc, char *argv[]) {
  wbArg_t args;
  int z_size;
  int y_size;
  int x_size;
  int inputLength, kernelLength;
  float *hostInput;
  float *hostKernel;
  float *hostOutput;
  float *deviceInput;
  float *deviceOutput;

  args = wbArg_read(argc, argv);

  // Import data
  hostInput = (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
  hostKernel =
      (float *)wbImport(wbArg_getInputFile(args, 1), &kernelLength);
  hostOutput = (float *)malloc(inputLength * sizeof(float));

  // First three elements are the input dimensions
  z_size = hostInput[0];
  y_size = hostInput[1];
  x_size = hostInput[2];
  wbLog(TRACE, "The input size is ", z_size, "x", y_size, "x", x_size);
  assert(z_size * y_size * x_size == inputLength - 3);
  assert(kernelLength == 27);

  // @@ Create local variables for accelerating computing
  int inputSize = x_size * y_size * z_size * sizeof(float); // or (inputLength - 3) * sizeof(float)
  int kernelSize = MASK_WIDTH * MASK_WIDTH * MASK_WIDTH * sizeof(float); // or kernelLength * sizeof(float)

  wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

  wbTime_start(GPU, "Doing GPU memory allocation");
  //@@ Allocate GPU memory here
  // Recall that inputLength is 3 elements longer than the input data
  // because the first three elements were the dimensions
  hipMalloc((void**) &deviceInput, inputSize);
  hipMalloc((void**) &deviceOutput, inputSize);
  wbTime_stop(GPU, "Doing GPU memory allocation");

  wbTime_start(Copy, "Copying data to the GPU");
  //@@ Copy input and kernel to GPU here
  // Recall that the first three elements of hostInput are dimensions and
  // do not need to be copied to the gpu
  hipMemcpy(deviceInput, hostInput+3, inputSize, hipMemcpyHostToDevice); // for normal memory
  hipMemcpyToSymbol(HIP_SYMBOL(deviceKernel), hostKernel, kernelSize); // for constant memory
  wbTime_stop(Copy, "Copying data to the GPU");

  wbTime_start(Compute, "Doing the computation on the GPU");
  //@@ Initialize grid and block dimensions here
  dim3 dimBlock(BLOCK_WIDTH, BLOCK_WIDTH, BLOCK_WIDTH); // note the padding
  dim3 dimGrid(ceil((float)x_size/TILE_WIDTH), ceil((float)y_size/TILE_WIDTH), ceil((float)z_size/TILE_WIDTH));
  //@@ Launch the GPU kernel here
  conv3d<<<dimGrid, dimBlock>>>(deviceInput, deviceOutput, z_size, y_size, x_size);
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Doing the computation on the GPU");

  wbTime_start(Copy, "Copying data from the GPU");
  //@@ Copy the device memory back to the host here
  // Recall that the first three elements of the output are the dimensions
  // and should not be set here (they are set below)
  hipMemcpy(hostOutput + 3, deviceOutput, inputSize, hipMemcpyDeviceToHost);
  wbTime_stop(Copy, "Copying data from the GPU");

  wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

  // Set the output dimensions for correctness checking
  hostOutput[0] = z_size;
  hostOutput[1] = y_size;
  hostOutput[2] = x_size;
  wbSolution(args, hostOutput, inputLength);

  // Free device memory
  hipFree(deviceInput);
  hipFree(deviceOutput);

  // Free host memory
  free(hostInput);
  free(hostOutput);
  return 0;
}