#include "hip/hip_runtime.h"
// MP 1
#include <wb.h>

__global__ void vecAdd(float *in1, float *in2, float *out, int len) {
  //@@ Insert code to implement vector addition here
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len)
    out[i] = in1[i] + in2[i];
}

int main(int argc, char **argv) {
  wbArg_t args;
  int inputLength;
  float *hostInput1;
  float *hostInput2;
  float *hostOutput;
  float *deviceInput1;
  float *deviceInput2;
  float *deviceOutput;

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostInput1 =
      (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
  hostInput2 =
      (float *)wbImport(wbArg_getInputFile(args, 1), &inputLength);
  hostOutput = (float *)malloc(inputLength * sizeof(float));
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
  int size = inputLength * sizeof(float);
  hipMalloc((void **) &deviceInput1, size);
  hipMalloc((void **) &deviceInput2, size);
  hipMalloc((void **) &deviceOutput, size);
  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here
  hipMemcpy(deviceInput1, hostInput1, size, hipMemcpyHostToDevice);
  hipMemcpy(deviceInput2, hostInput2, size, hipMemcpyHostToDevice);
  wbTime_stop(GPU, "Copying input memory to the GPU.");
  
  //@@ Initialize the grid and block dimensions here
  dim3 DimGrid(ceil(inputLength/256.0), 1, 1);
  dim3 DimBlock(256, 1, 1);
  
  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Launch the GPU Kernel here
  vecAdd<<<DimGrid, DimBlock>>>(deviceInput1, deviceInput2, deviceOutput, inputLength);
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");
  
  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostOutput, deviceOutput, size, hipMemcpyDeviceToHost);
  wbTime_stop(Copy, "Copying output memory to the CPU");
  
  wbTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
  hipFree(deviceInput1);
  hipFree(deviceInput2);
  hipFree(deviceOutput);
  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostOutput, inputLength);

  free(hostInput1);
  free(hostInput2);
  free(hostOutput);
  return 0;
}